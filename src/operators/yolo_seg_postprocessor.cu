#include <hip/hip_runtime.h>

// CUDA kernel for GPU-based YOLO postprocessing
__global__ void yolo_postprocess_kernel(
    const float* predictions,
    float* output_boxes,
    float* output_scores,
    int* output_class_ids,
    int* valid_detections,
    int num_detections,
    int feature_size,
    float confidence_threshold,
    int num_classes
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_detections) return;
    
    // Calculate offset for this detection
    int offset = idx * feature_size;
    
    // Extract basic features (using coalesced memory access)
    float x = predictions[offset + 0];
    float y = predictions[offset + 1];
    float w = predictions[offset + 2];
    float h = predictions[offset + 3];
    float confidence = predictions[offset + 4];
    float class_id_float = predictions[offset + 5];
    
    // Early exit for invalid detections
    int class_id = static_cast<int>(class_id_float);
    if (class_id < 0 || class_id >= num_classes || confidence < confidence_threshold) {
        valid_detections[idx] = 0;
        return;
    }
    
    // Convert to xyxy format (clamp to valid range)
    float x1 = fmaxf(0.0f, x - w / 2.0f);
    float y1 = fmaxf(0.0f, y - h / 2.0f);
    float x2 = fminf(1.0f, x + w / 2.0f);
    float y2 = fminf(1.0f, y + h / 2.0f);
    
    // Validate box dimensions
    if (x2 <= x1 || y2 <= y1) {
        valid_detections[idx] = 0;
        return;
    }
    
    // Store results
    output_boxes[idx * 4 + 0] = x1;
    output_boxes[idx * 4 + 1] = y1;
    output_boxes[idx * 4 + 2] = x2;
    output_boxes[idx * 4 + 3] = y2;
    output_scores[idx] = confidence;
    output_class_ids[idx] = class_id;
    valid_detections[idx] = 1;
}

// CUDA kernel for Non-Maximum Suppression (optimized)
__global__ void nms_kernel(
    const float* boxes,
    const float* scores,
    const int* class_ids,
    int* keep_flags,
    int num_detections,
    float iou_threshold
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_detections) return;
    
    if (keep_flags[idx] == 0) return; // Already suppressed
    
    float x1_i = boxes[idx * 4 + 0];
    float y1_i = boxes[idx * 4 + 1];
    float x2_i = boxes[idx * 4 + 2];
    float y2_i = boxes[idx * 4 + 3];
    float score_i = scores[idx];
    int class_i = class_ids[idx];
    
    // Calculate area_i once
    float area_i = (x2_i - x1_i) * (y2_i - y1_i);
    
    // Compare with all other detections
    for (int j = idx + 1; j < num_detections; ++j) {
        if (keep_flags[j] == 0) continue; // Already suppressed
        if (class_ids[j] != class_i) continue; // Different class
        
        float x1_j = boxes[j * 4 + 0];
        float y1_j = boxes[j * 4 + 1];
        float x2_j = boxes[j * 4 + 2];
        float y2_j = boxes[j * 4 + 3];
        float score_j = scores[j];
        
        // Calculate IoU (optimized)
        float x1_inter = fmaxf(x1_i, x1_j);
        float y1_inter = fmaxf(y1_i, y1_j);
        float x2_inter = fminf(x2_i, x2_j);
        float y2_inter = fminf(y2_i, y2_j);
        
        // Early exit if no overlap
        if (x2_inter <= x1_inter || y2_inter <= y1_inter) continue;
        
        float inter_area = (x2_inter - x1_inter) * (y2_inter - y1_inter);
        float area_j = (x2_j - x1_j) * (y2_j - y1_j);
        float union_area = area_i + area_j - inter_area;
        
        // Avoid division by zero
        if (union_area <= 0.0f) continue;
        
        float iou = inter_area / union_area;
        
        // Suppress the detection with lower score
        if (iou > iou_threshold) {
            if (score_i > score_j) {
                keep_flags[j] = 0;
            } else {
                keep_flags[idx] = 0;
                break;
            }
        }
    }
}

// C wrapper functions for calling from C++ code
extern "C" {
    void launch_yolo_postprocess_kernel(
        const float* predictions,
        float* output_boxes,
        float* output_scores,
        int* output_class_ids,
        int* valid_detections,
        int num_detections,
        int feature_size,
        float confidence_threshold,
        int num_classes,
        hipStream_t stream = 0
    ) {
        int block_size = 256;
        int grid_size = (num_detections + block_size - 1) / block_size;
        
        yolo_postprocess_kernel<<<grid_size, block_size, 0, stream>>>(
            predictions,
            output_boxes,
            output_scores,
            output_class_ids,
            valid_detections,
            num_detections,
            feature_size,
            confidence_threshold,
            num_classes
        );
    }
    
    void launch_nms_kernel(
        const float* boxes,
        const float* scores,
        const int* class_ids,
        int* keep_flags,
        int num_detections,
        float iou_threshold,
        hipStream_t stream = 0
    ) {
        int block_size = 256;
        int grid_size = (num_detections + block_size - 1) / block_size;
        
        nms_kernel<<<grid_size, block_size, 0, stream>>>(
            boxes,
            scores,
            class_ids,
            keep_flags,
            num_detections,
            iou_threshold
        );
    }
} 